#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define V 8
#define E 11
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;

typedef struct
{
	int u;
	int v;
} Edge;


typedef struct 
{
	int title;
	boolean visited;	
}Vertex;


__device__ __host__ int findEdge(Vertex u, Vertex v, Edge *edges, int *weights)
{

	int i;
	for(i = 0; i < E; i++)
	{
		if((edges[i].u == u.title && edges[i].v == v.title) || (edges[i].v == u.title && edges[i].u == v.title))
		{
			return weights[i];
		}
	}

	return MAX_WEIGHT;
}


__global__ void initVertices(Vertex *vertices, Edge *edges, int* weights, int* length, int* updateLength, Vertex root){
    
    int i = threadIdx.x;
    
    if(vertices[i].title != root.title)
		{
			length[(int)vertices[i].title] = findEdge(root, vertices[i], edges, weights);
			updateLength[vertices[i].title] = length[(int)vertices[i].title];		
		}
		else{
			vertices[i].visited = TRUE;
		}
}


__global__ void findVertex(Vertex *vertices, Edge *edges, int *weights, int *length, int *updateLength, int* path)
{

	int u = threadIdx.x;

	if(vertices[u].visited == FALSE)
	{

		vertices[u].visited = TRUE;

		int v;
		for(v = 0; v < V; v++)
		{	
			int weight = findEdge(vertices[u], vertices[v], edges, weights);

			if(weight < MAX_WEIGHT)
			{	
				
				if(updateLength[v] > length[u] + weight)
				{
			        path[v] = u;
					    updateLength[v] = length[u] + weight;
				}
			}
		}
	}
}


__global__ void updatePaths(Vertex *vertices, int *length, int *updateLength)
{
	int u = threadIdx.x;
	if(length[u] > updateLength[u])
	{
		length[u] = updateLength[u];
		vertices[u].visited = FALSE;
	}

	updateLength[u] = length[u];
}



void printShortestPath(int *array, int dest, int *req_path, int count)
{
	printf("Shortest Path to Vertex: %d is %d\nPATH: ", dest, array[dest]);
	for(int i=count-1;i>=0;i--)
    {
        printf("%d-->", req_path[i]);
    }
    printf("%d", dest);
}



int main(void)
{
    
	Vertex *vertices;	
	Edge *edges;

	
	int *weights;
  	int *path;
  	int *len, *updateLength;
	
	Vertex *d_V;
	Edge *d_E;
	int *d_W;
	int *d_L;
	int *d_C, *d_P;
	
	int sizeV = sizeof(Vertex) * V;
	int sizeE = sizeof(Edge) * E;
	int size = V * sizeof(int);
	
	float runningTime;
	hipEvent_t timeStart, timeEnd;
	
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);

	vertices = (Vertex *)malloc(sizeV);
	edges = (Edge *)malloc(sizeE);
	weights = (int *)malloc(E* sizeof(int));
  	path = (int *)malloc(V*sizeof(int));
	len = (int *)malloc(size);
	updateLength = (int *)malloc(size);

	Edge ed[E] = {{0, 4}, {0, 6}, {0,2}, {4,6}, {4,7}, {0, 7}, {7, 3}, {3, 1}, {2,5}, {2, 1}, {5,3}};
	int w[E] = {10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10};
	
	hipMalloc((void**)&d_V, sizeV);
	hipMalloc((void**)&d_E, sizeE);
	hipMalloc((void**)&d_W, E * sizeof(int));
	hipMalloc((void**)&d_L, size);
	hipMalloc((void**)&d_C, size);
  	hipMalloc((void**)&d_P, size);

	Vertex root = {7, FALSE};
	root.visited = TRUE;
	len[root.title] = 0;
	updateLength[root.title] = 0;

	int dest = 0;

	int i = 0;
	for(i = 0; i < V; i++)
	{
		Vertex a = { i , FALSE};
		vertices[i] = a;
    	path[i] = root.title;
	}

	for(i = 0; i < E; i++)
	{
		edges[i] = ed[i];
		weights[i] = w[i];
	}

    
	hipMemcpy(d_V, vertices, sizeV, hipMemcpyHostToDevice);
	hipMemcpy(d_E, edges, sizeE, hipMemcpyHostToDevice);
	hipMemcpy(d_W, weights, E * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_L, len, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, size, hipMemcpyHostToDevice);
  	hipMemcpy(d_P, path, size, hipMemcpyHostToDevice);
    
    
 	initVertices<<<1, V>>>(d_V, d_E, d_W, d_L, d_C, root);
	
  	hipMemcpy(len, d_L, size, hipMemcpyDeviceToHost);
  	hipMemcpy(updateLength, d_C, size, hipMemcpyDeviceToHost);

	hipEventRecord(timeStart, 0);
	
	hipMemcpy(d_L, len, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, size, hipMemcpyHostToDevice);
	
	for(i = 1; i < V; i++)
	{
		findVertex<<<1, V>>>(d_V, d_E, d_W, d_L, d_C, d_P);
		updatePaths<<<1,V>>>(d_V, d_L, d_C);
	}	
	
	
	hipEventRecord(timeEnd, 0);
	hipEventSynchronize(timeEnd);
	hipEventElapsedTime(&runningTime, timeStart, timeEnd);

	hipMemcpy(len, d_L, size, hipMemcpyDeviceToHost);
  	hipMemcpy(path, d_P, size, hipMemcpyDeviceToHost);
    
    int req_path[V];
    int temp = dest;
    int count = 0;
    while(temp!=root.title)
    {
    	req_path[count++] = path[temp];
        temp = path[temp];
    }
    
    printShortestPath(len, dest, req_path, count);

	printf("Running Time: %f ms\n", runningTime);

	free(vertices);
	free(edges);
	free(weights);
	free(len);
	free(updateLength);
	hipFree(d_V);
	hipFree(d_E);
	hipFree(d_W);
	hipFree(d_L);
	hipFree(d_C);
	hipFree(d_P);
	hipEventDestroy(timeStart);
	hipEventDestroy(timeEnd);
}